#include "hip/hip_runtime.h"
#include "utilities_cuda.hpp"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <>



__global__ void updateIntensities(float* intensities, int num_squares) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_squares) {
        intensities[idx] = intensities[idx] + 0.001f;
    }
}

int main() {

    float* d_intensities;
    hipMalloc(&d_intensities, GRID_WIDTH * GRID_HEIGHT * sizeof(float));

    // Initialize GLFW
    if (!glfwInit())
    {
        std::cout << "Failed to initialize GLFW" << std::endl;
        return -1;
    }

    // Configure GLFW
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    #ifdef __APPLE__
        glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
    #endif

    // Create a window
    GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "OpenGL Squares", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebufferSizeCallback);

    // Initialize GLEW
    if (glewInit() != GLEW_OK)
    {
        std::cout << "Failed to initialize GLEW" << std::endl;
        return -1;
    }

    // Vertex data for a single square
    float vertices[] = {
        0.5f,  0.5f,
        0.5f, -0.5f,
        -0.5f, -0.5f,
        -0.5f,  0.5f
    };

    unsigned int indices[] = {
        0, 1, 3,
        1, 2, 3
    };

    // Set up VAO and VBO
    unsigned int VAO, VBO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);

    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

    glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 2 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(0);

    // Create intensity VBO and register with CUDA
    GLuint intensityVBO;
    glGenBuffers(1, &intensityVBO);
    glBindBuffer(GL_ARRAY_BUFFER, intensityVBO);
    glBufferData(GL_ARRAY_BUFFER, GRID_WIDTH * GRID_HEIGHT * sizeof(float), NULL, GL_DYNAMIC_DRAW);

    hipGraphicsResource* cuda_intensity_resource;
    hipGraphicsGLRegisterBuffer(&cuda_intensity_resource, intensityVBO, cudaGraphicsMapFlagsWriteDiscard);

    // Create and compile shaders
    unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);

    unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);

    // Create shader program
    unsigned int shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);

    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);

    unsigned int intensityLoc = glGetUniformLocation(shaderProgram, "intensity");
   // Render loop
    while(!glfwWindowShouldClose(window)) {
        // Clear the screen with black color
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // Map OpenGL buffer to CUDA
        float* d_mapped_intensities;
        size_t num_bytes;
        hipGraphicsMapResources(1, &cuda_intensity_resource, 0);
        hipGraphicsResourceGetMappedPointer((void**)&d_mapped_intensities, &num_bytes, cuda_intensity_resource);

        // Update intensities using CUDA
        int threadsPerBlock = 256;
        int blocksPerGrid = (GRID_WIDTH * GRID_HEIGHT + threadsPerBlock - 1) / threadsPerBlock;
        updateIntensities<<<blocksPerGrid, threadsPerBlock>>>(d_mapped_intensities, GRID_WIDTH * GRID_HEIGHT);

        // Unmap buffer
        hipGraphicsUnmapResources(1, &cuda_intensity_resource, 0);

        // Use the updated intensities in OpenGL
        glBindBuffer(GL_ARRAY_BUFFER, intensityVBO);
        glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, sizeof(float), (void*)0);
        glEnableVertexAttribArray(1);

        // Use shader program
        glUseProgram(shaderProgram);

        // Draw squares
        glBindVertexArray(VAO);
        float spacing = 0.2f; // Space between squares
        float totalWidth = (GRID_WIDTH - 1) * (1 + spacing);
        float totalHeight = (GRID_HEIGHT - 1) * (1 + spacing);
        for (int i = 0; i < GRID_WIDTH; i++) {
            for (int j = 0; j < GRID_HEIGHT; j++) {
                int index = i * GRID_HEIGHT + j;
                
                glm::mat4 model = glm::mat4(1.0f);
                model = glm::translate(model, glm::vec3(
                    (i * (1 + spacing) - totalWidth / 2) / GRID_WIDTH,
                    (j * (1 + spacing) - totalHeight / 2) / GRID_HEIGHT,
                    0.0f
                ));
                model = glm::scale(model, glm::vec3(1.0f / (GRID_WIDTH * (1 + spacing)), 
                                                    1.0f / (GRID_HEIGHT * (1 + spacing)), 
                                                    1.0f));
                
                unsigned int modelLoc = glGetUniformLocation(shaderProgram, "model");
                glUniformMatrix4fv(modelLoc, 1, GL_FALSE, glm::value_ptr(model));
                
                // Set the intensity for this square
                float intensity = 1.0;
                glUniform1f(intensityLoc, intensity);
                
                glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
            }
        }

        // Swap buffers and poll events
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    // Clean up
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glDeleteProgram(shaderProgram);
    glfwTerminate();
    hipFree(d_intensities);
    hipGraphicsUnregisterResource(cuda_intensity_resource);
    return 0;
}